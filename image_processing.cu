#include "hip/hip_runtime.h"
#include "image_processing.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <dirent.h>
#include <string.h>

// CUDA kernel to convert RGB to grayscale
__global__ void grayscaleKernel(unsigned char *d_in, unsigned char *d_out, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * 3; // RGB input
        int out_idx = y * width + x;    // Grayscale output
        unsigned char r = d_in[idx];
        unsigned char g = d_in[idx + 1];
        unsigned char b = d_in[idx + 2];
        d_out[out_idx] = (unsigned char)(0.299f * r + 0.587f * g + 0.114f * b);
    }
}

// Load a PPM image (P6 format)
Image* loadPPM(const char* filename) {
    FILE *fp = fopen(filename, "rb");
    if (!fp) return nullptr;

    char buffer[16];
    fscanf(fp, "%s\n", buffer); // P6
    int width, height, maxval;
    fscanf(fp, "%d %d\n%d\n", &width, &height, &maxval);

    Image* img = (Image*)malloc(sizeof(Image));
    img->width = width;
    img->height = height;
    img->data = (unsigned char*)malloc(width * height * 3);
    fread(img->data, 1, width * height * 3, fp);
    fclose(fp);
    return img;
}

// Save a PPM image (grayscale as RGB for simplicity)
void savePPM(const char* filename, Image* img) {
    FILE *fp = fopen(filename, "wb");
    fprintf(fp, "P6\n%d %d\n255\n", img->width, img->height);
    fwrite(img->data, 1, img->width * img->height * 3, fp);
    fclose(fp);
}

void freeImage(Image* img) {
    free(img->data);
    free(img);
}

void processImages(const char* inputDir, const char* outputDir, int numImages) {
    DIR *dir = opendir(inputDir);
    if (!dir) {
        printf("Error opening input directory\n");
        return;
    }

    struct dirent *entry;
    int processed = 0;
    while ((entry = readdir(dir)) != nullptr && processed < numImages) {
        if (strstr(entry->d_name, ".ppm") == nullptr) continue;

        char inputPath[256], outputPath[256];
        snprintf(inputPath, sizeof(inputPath), "%s/%s", inputDir, entry->d_name);
        snprintf(outputPath, sizeof(outputPath), "%s/%s", outputDir, entry->d_name);

        Image* img = loadPPM(inputPath);
        if (!img) continue;

        // Allocate device memory
        unsigned char *d_in, *d_out;
        size_t rgbSize = img->width * img->height * 3;
        size_t graySize = img->width * img->height;
        hipMalloc(&d_in, rgbSize);
        hipMalloc(&d_out, graySize);

        // Copy input to device
        hipMemcpy(d_in, img->data, rgbSize, hipMemcpyHostToDevice);

        // Launch kernel
        dim3 block(16, 16);
        dim3 grid((img->width + block.x - 1) / block.x, (img->height + block.y - 1) / block.y);
        grayscaleKernel<<<grid, block>>>(d_in, d_out, img->width, img->height);

        // Copy result back and convert to RGB for PPM
        unsigned char *h_out = (unsigned char*)malloc(graySize);
        hipMemcpy(h_out, d_out, graySize, hipMemcpyDeviceToHost);
        for (int i = 0; i < img->width * img->height; i++) {
            img->data[i * 3] = h_out[i];
            img->data[i * 3 + 1] = h_out[i];
            img->data[i * 3 + 2] = h_out[i];
        }
        free(h_out);

        // Save output
        savePPM(outputPath, img);

        // Cleanup
        hipFree(d_in);
        hipFree(d_out);
        freeImage(img);
        processed++;
        printf("Processed: %s\n", entry->d_name);
    }
    closedir(dir);
}
